#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<float.h>
#define CUDA_CALL(x) do { hipError_t err=(x); \
	if(err!=hipSuccess) { \
	printf("Error %s at %s: %d",hipGetErrorString(err),__FILE__,__LINE__);\
	return EXIT_FAILURE;}} while(0)  



#define W 64 // Input DIM
#define D 3   // Input and Kernel Depth
#define T 5  // Kernel DIM
#define N 128 // Number of kernels
#define TILE_W 8 //input tile width
#define n1 3 //Range for weights(log2) from INQ
#define n2 1 //where n1 > n2
#define BAND 3 // range for weights
#define STRIDE_LENGTH 1 //STRIDE_LENGTH
#define OWS (W- T + 1) // Output DIM
#define OW (((W - T)/STRIDE_LENGTH) + 1) //output width

__global__ void cudaConvolve(float* output,  int*  kernel, unsigned char *matrix){
/*

one block loads its required tile from the matrix collaboritively
and calculates the values for the number of kernels equalling to blockdim.x

*/		
	__shared__ float shmatrix[TILE_W+T-1][TILE_W+T-1][D];
	__shared__ int shkernel[D][T][T][D][2];	

	float Sum[BAND];

	float ds=0.0;
	
	long i=0,j=0,k=0,m=0;
	
	long ty = threadIdx.y;
	long tx = threadIdx.x;
	long tz = threadIdx.z;
	
	long z = blockIdx.z*TILE_W+tz;
	long y = blockIdx.y*TILE_W+ty;
	long x = blockIdx.x*blockDim.x + tx;

	//kernel contains the abs log of weight and the sign
		
	if (ty<T && tz<T){
		for(k=0;k<D;++k){
			shkernel[k][tz][ty][tx][0] = kernel[(x-tx+k)*2*D*T*T + tz*2*D*T+ ty*2*D+ 2*tx];
			shkernel[k][tz][ty][tx][1] = kernel[(x-tx+k)*2*D*T*T + tz*2*D*T+ ty*2*D+ 2*tx+1];
		}	
	}
	__syncthreads();	

	if ( z>=0 && z <W && y>=0 && y <W){
		shmatrix[tz][ty][tx] = matrix[z*D*W + y* D+ tx];
	}
	else
		shmatrix[tz][ty][tx] = 0.0f;


	__syncthreads();
	
	//sum array stores the sum of matrix element sharing the same weights
	for(m=0;m<BAND;m++){
		Sum[m]=0.0;
	}

	if(y%STRIDE_LENGTH == 0 && z%STRIDE_LENGTH == 0){
		if (ty<TILE_W && tz<TILE_W){
			for(k=0;k<D;++k){
				for(i=0;i<T;++i){
					for(j=0;j<T;++j){
						if(shkernel[tx][i][j][k][1] > 0){
							Sum[shkernel[tx][i][j][k][0] - n2] += shmatrix[i+tz][ty+j][k];
						}
						if(shkernel[tx][i][j][k][1] < 0){
							Sum[shkernel[tx][i][j][k][0] - n2] -= shmatrix[i+tz][ty+j][k];	
						}
					}
				}
			}
		}	

		for(m=0;m<BAND;m++){
			if(m + n2 > 0){
				ds+=Sum[m]*(1<<(m + n2));
			}
			else{
				ds+=Sum[m]/(1<<((-1)*(m + n2)));	
			}
		}
		__syncthreads();
		
		if (z<OWS && y<OWS && ty<TILE_W && tz<TILE_W){	
			output[x*OW*OW + (z/STRIDE_LENGTH)*OW + (y/STRIDE_LENGTH)] = ds;
		}
	}

}

void fillMatrix(unsigned char *matrix){

unsigned char (*m)[W][D]=(unsigned char (*)[W][D])matrix;

for(int i=0;i<W;i++){
	for(int j=0;j<W;j++){
		for(int k=0;k<D;k++){
			m[i][j][k]=(i*j+j*k+i*k+i*2+j*3+k*4)%255;
				}
			}
		}
}



void fillKernel(int *kernel){

int (*t)[T][T][D][2]=(int (*)[T][T][D][2])kernel;

for(int i=0;i<N;i++){
	for(int j=0;j<T;j++){
		for(int k=0;k<T;k++){
			for(int l=0;l<D;l++){
				t[i][j][k][l][0]=((i+j+T+D)%n1 + n2);
				t[i][j][k][l][1]=(pow(-1,i+j));
			}
		}
	}
}
}




void printtofile(float *m){

	const char *fname = "GPU_TAST";
	FILE *f = fopen(fname, "w");

	float (*mat)[OW][OW]=(float (*)[OW][OW])m;		

	for(unsigned i=0; i < N; i++) {
		for(unsigned j=0; j < OW; j++){
			for(unsigned k=0;k<OW;k++){
				fprintf(f,"%4.4f ", mat[i][j][k]);
			}
			fprintf(f, "\n");
		}
		fprintf(f,"\n");
	}
	fclose(f);
}



int main()
{

	unsigned char *matrix=(unsigned char*)malloc(sizeof(unsigned char)*W*W*D);
	int *kernel=(int*)malloc(sizeof(int)*T*T*D*N*2);
	float *output=(float *)malloc(sizeof(float)*N*OW*OW);


	fillMatrix(matrix);
	fillKernel(kernel);


	unsigned char *Dmatrix;
	hipMalloc(&Dmatrix,sizeof(unsigned char)*W*W*D);
	int *Dkernel;
	hipMalloc(&Dkernel,sizeof(int)*N*T*T*D*2);
	float *Doutput;
	hipMalloc(&Doutput,sizeof(float)*N*OW*OW);

	hipMemcpy(Dmatrix, matrix, sizeof(unsigned char)*W*W*D,hipMemcpyHostToDevice);
	hipMemcpy(Dkernel, kernel, sizeof(int)*T*T*D*N*2,hipMemcpyHostToDevice);


	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;
	hipEventRecord(start,0);

	//Kernel Call
	dim3 threads(D,TILE_W+T-1,TILE_W+T-1);
	dim3 blocks(N/D, (W+TILE_W-1)/TILE_W , (W+TILE_W-1)/TILE_W );

	cudaConvolve<<< blocks, threads >>>(Doutput, Dkernel, Dmatrix);
	CUDA_CALL(hipGetLastError());
	
	hipDeviceSynchronize();


	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f\n",milliseconds);


	hipMemcpy(output, Doutput, sizeof(float)*N*OW*OW,hipMemcpyDeviceToHost);

	//printtofile(output);

}
