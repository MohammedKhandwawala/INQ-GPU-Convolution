//Strided convolution output stationary 
//In this program , INQ weight sharing property is used weights are quatized 
//each thread computes one output element. so the matrix elements with common 
//weights are added up then just multiplied once.  

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#define CUDA_CALL(x) do { hipError_t err=(x); \
	if(err!=hipSuccess) { \
	printf("Error %s at %s: %d",hipGetErrorString(err),__FILE__,__LINE__);\
	return EXIT_FAILURE;}} while(0)  
#define W 64 // Input DIM
#define D 4  // Input and Kernel Depth
#define T 5  // Kernel DIM
#define N 2 // Number of kernels
#define TILE_W 16 //output tile size
#define n1 3 //range for weights in INQ
#define n2 1 //n1 > n2
#define BAND 3
#define STRIDE_LENGTH 1//stride length		
#define OWS (W- T + 1) // Output DIM
#define OW (((W - T)/STRIDE_LENGTH) + 1)

void fillMatrix(unsigned char *matrix){

unsigned char (*m)[W][D]=(unsigned char (*)[W][D])matrix;

for(int i=0;i<W;i++){
	for(int j=0;j<W;j++){
		for(int k=0;k<D;k++){
			m[i][j][k]=(i*j+j*k+i*k+i*2+j*3+k*4)%255;
				}
			}
		}
}

void fillKernel(int *kernel){

int (*t)[T][T][D][2]=(int (*)[T][T][D][2])kernel;

for(int i=0;i<N;i++){
	for(int j=0;j<T;j++){
		for(int k=0;k<T;k++){
			for(int l=0;l<D;l++){
				t[i][j][k][l][0]=((i+j+T+D)%n1 + n2);
				t[i][j][k][l][1]=(pow(-1,i+j));
			}
		}
	}
}
}



void printtofile(float *m){

	const char *fname = "GPU_TAST";
	FILE *f = fopen(fname, "w");

	float (*mat)[OW][OW]=(float (*)[OW][OW])m;		

	for(unsigned i=0; i < N; i++) {
		for(unsigned j=0; j < OW; j++){
			for(unsigned k=0;k<OW;k++){
				fprintf(f,"%4.4f ", mat[i][j][k]);
			}
			fprintf(f, "\n" );
		}
		fprintf(f,"\n");
	}
	fclose(f);
}

__global__ void conv(unsigned char* Dm, int* Dk, float* Do)
{
	__shared__ int ker[2*T*T*D];
	__shared__ unsigned char tile[(TILE_W)*(TILE_W)*D];
	int tx=blockDim.x*blockIdx.x+threadIdx.x;
	int ty=blockDim.y*blockIdx.y+threadIdx.y;
	int bz=blockIdx.z;
	int zk=bz*T*T*D;
	int ym,xm;
	    /* each thread computes one elemement in the output matrix 
	       kernel conntains log2 of abs of weights and sign
	    */
		for(int d=0;d<D;d++)
		{
			if(threadIdx.x<T&&threadIdx.y<T){
				ker[threadIdx.y*2*T*D+threadIdx.x*2*D+2*d]=Dk[2*zk+threadIdx.y*2*T*D+threadIdx.x*2*D+2*d];
				ker[threadIdx.y*2*T*D+threadIdx.x*2*D+2*d+1]=Dk[2*zk+threadIdx.y*2*T*D+threadIdx.x*2*D+2*d + 1];
			}
		}
		//__syncthreads();
		for(int d=0;d<D;d++)
		{
			ym=ty*W*D;
			xm=tx*D;
			tile[threadIdx.y*(TILE_W)*D+threadIdx.x*D+d]=Dm[ym+xm+d];
			if((tx+(TILE_W - T + 1))<W&&(threadIdx.x+(TILE_W - T + 1))<(TILE_W))
			{
				ym=ty*W*D;
				xm=(tx+(TILE_W - T + 1))*D;
				tile[threadIdx.y*(TILE_W)*D+(threadIdx.x+(TILE_W - T + 1))*D+d]=Dm[ym+xm+d];
			}
			if((ty+(TILE_W - T + 1))<W&&(threadIdx.y+(TILE_W - T + 1))<(TILE_W))
			{
				ym=(ty+(TILE_W - T + 1))*W*D;
				xm=(tx)*D;
				tile[(threadIdx.y+(TILE_W - T + 1))*(TILE_W)*D+(threadIdx.x)*D+d]=Dm[ym+xm+d];
			}
			if(((ty+(TILE_W - T + 1))<W&&(threadIdx.y+(TILE_W - T + 1))<(TILE_W))&&((tx+(TILE_W - T + 1))<W&&(threadIdx.x+(TILE_W - T + 1))<(TILE_W)))
			{
				ym=(ty+(TILE_W - T + 1))*W*D;
				xm=(tx+(TILE_W - T + 1))*D;
				tile[(threadIdx.y+(TILE_W - T + 1))*(TILE_W)*D+(threadIdx.x+(TILE_W - T + 1))*D+d]=Dm[ym+xm+d];
			}
		}
	__syncthreads();

	//matirx multiplication , thread computing one element
	//sum array is store in thread array which stores the sum for each weight
	if(ty%STRIDE_LENGTH == 0 && tx%STRIDE_LENGTH == 0)
	{
		float psum=0.0;
		float sum[BAND];

		for(int i=0; i < BAND; i++){
			sum[i] = 0.0;	
		}

		for(int i=0;i<T;i++)
		{
			int yk1=i*2*T*D;
			int ym1=(threadIdx.y+i)*(TILE_W)*D;
			for(int j=0;j<T;j++)
			{
				int xk1=j*2*D;
				int xm1=(threadIdx.x+j)*D;
				for(int d=0;d<D;d++){
					if(ker[yk1+xk1+2*d+1] > 0){
						sum[ker[yk1+xk1+2*d] - n2]+=tile[ym1+xm1+d];
					}
					else{
						sum[ker[yk1+xk1+2*d] - n2]-=tile[ym1+xm1+d];
					}
				}
			}
		}	
		for(int i =0;i < BAND; i++){
			if(i+n2>0){
				psum+=sum[i]*(1<<(i + n2));
			}
			else{
				psum+=sum[i]/(1<<((-1)*(i + n2)));
			}
		}
		if(tx<OWS&&ty<OWS){ 
			Do[bz*OW*OW+(ty/STRIDE_LENGTH)*OW+(tx/STRIDE_LENGTH)]=psum;
		}
	}
}

int main()
{

	//allocating memory on the host
	unsigned char *matrix=(unsigned char*)malloc(sizeof(unsigned char)*W*W*D);
	int *kernel=(int*)malloc(sizeof(int)*2*T*T*D*N);
	float *output=(float *)malloc(sizeof(float)*N*OW*OW);

	//filling the matrix and the kernel
	fillMatrix(matrix);
	fillKernel(kernel);

	//allocating memory on the GPU
	unsigned char *Dmatrix;hipMalloc(&Dmatrix,sizeof(unsigned char)*W*W*D);
	int *Dkernel;hipMalloc(&Dkernel,sizeof(int)*2*N*T*T*D);
	float *Doutput;hipMalloc(&Doutput,sizeof(float)*N*OW*OW);
	int blockdimx=(TILE_W - T + 1);
	int blockdimy=(TILE_W - T + 1);
	int griddimz=N;
	int griddimy=(OWS+blockdimx-1)/blockdimx;
	int griddimx=(OWS+blockdimy-1)/blockdimy;
	dim3 blocks(griddimx, griddimy, griddimz);
	dim3 thrds_per_block(blockdimx, blockdimy);

	//copying kernel and the matrix to the GPU
	hipMemcpy(Dmatrix, matrix, sizeof(unsigned char)*W*W*D,hipMemcpyHostToDevice);
	hipMemcpy(Dkernel, kernel, sizeof(int)*2*T*T*D*N,hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;
	hipEventRecord(start,0);

	//cuda kernel call
	conv<<<blocks,thrds_per_block>>>(Dmatrix, Dkernel, Doutput);
	CUDA_CALL(hipGetLastError());
	
	hipDeviceSynchronize();

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f\n",milliseconds);


	hipMemcpy(output, Doutput, sizeof(float)*N*OW*OW,hipMemcpyDeviceToHost);

	//Use print_matrix_to_file function only 
	printtofile(output);

}
