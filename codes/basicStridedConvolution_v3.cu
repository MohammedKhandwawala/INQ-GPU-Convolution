/*	Normal strided convolution output tiled code to benchmark
	stridedConvolution_v3.cu .
*/ 
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#define CUDA_CALL(x) do { hipError_t err=(x); \
	if(err!=hipSuccess) { \
	printf("Error %s at %s: %d",hipGetErrorString(err),__FILE__,__LINE__);\
	return EXIT_FAILURE;}} while(0)  
#define W 32 // Input DIM
#define D 4  // Input and Kernel Depth
#define T 5  // Kernel DIM
#define N 2 // Number of kernels
#define TILE_W 16 //Tile Size
#define n1 3 //range of weights from INQ
#define n2 1 //n1>n2
#define BAND 3 
#define STRIDE_LENGTH 1		
#define OWS (W- T + 1) // Output DIM
#define OW (((W - T)/STRIDE_LENGTH) + 1) //output DIM


void fillMatrix(unsigned char *matrix){

unsigned char (*m)[W][D]=(unsigned char (*)[W][D])matrix;

for(int i=0;i<W;i++){
	for(int j=0;j<W;j++){
		for(int k=0;k<D;k++){
			m[i][j][k]=(i*j+j*k+i*k+i*2+j*3+k*4)%255;
				}
			}
		}
}


void fillKernel(float *kernel){

float (*t)[T][T][D]=(float (*)[T][T][D])kernel;

for(int i=0;i<N;i++){
	for(int j=0;j<T;j++){
		for(int k=0;k<T;k++){
			for(int l=0;l<D;l++){
				if(((i+j+T+D)%n1 + n2) >= 0){
					t[i][j][k][l]=(1<<((i+j+T+D)%n1 + n2));//*(pow(-1,i+j));
				}
				if(((i+j+T+D)%n1 + n2) < 0){
					t[i][j][k][l]=(1.0/(1<<(-1*((i+j+T+D)%n1 + n2))));//*(pow(-1,i+j));
				}
			}
		}
	}
}
}


void printtofile(float *m){

	const char *fname = "GPU_TAST";
	FILE *f = fopen(fname, "w");

	float (*mat)[OW][OW]=(float (*)[OW][OW])m;		

	for(unsigned i=0; i < N; i++) {
		for(unsigned j=0; j < OW; j++){
			for(unsigned k=0;k<OW;k++){
				fprintf(f,"%4.4f ", mat[i][j][k]);
			}
			fprintf(f, "\n" );
		}
		fprintf(f,"\n");
	}
	fclose(f);
}

__global__ void conv(unsigned char* Dm, float* Dk, float* Do)
{
	/*  
	    each thread computes one output element
		Output Stationary
	*/

	__shared__ float ker[T*T*D];
	__shared__ unsigned char tile[(TILE_W)*(TILE_W)*D];

	int tx=blockDim.x*blockIdx.x+threadIdx.x;
	int ty=blockDim.y*blockIdx.y+threadIdx.y;
	int bz=blockIdx.z;
	int zk=bz*T*T*D;
	int ym,xm;
		//copying kernel to the shared memory
		for(int d=0;d<D;d++)
		{
			if(threadIdx.x<T&&threadIdx.y<T){
				ker[threadIdx.y*T*D+threadIdx.x*D+d]=Dk[zk+threadIdx.y*T*D+threadIdx.x*D+d];			}
		}
		//__syncthreads();
		for(int d=0;d<D;d++)
		{
			ym=ty*W*D;
			xm=tx*D;
			tile[threadIdx.y*(TILE_W)*D+threadIdx.x*D+d]=Dm[ym+xm+d];
			if((tx+(TILE_W - T + 1))<W&&(threadIdx.x+(TILE_W - T + 1))<(TILE_W))
			{
				ym=ty*W*D;
				xm=(tx+(TILE_W - T + 1))*D;
				tile[threadIdx.y*(TILE_W)*D+(threadIdx.x+(TILE_W - T + 1))*D+d]=Dm[ym+xm+d];
			}
			if((ty+(TILE_W - T + 1))<W&&(threadIdx.y+(TILE_W - T + 1))<(TILE_W))
			{
				ym=(ty+(TILE_W - T + 1))*W*D;
				xm=(tx)*D;
				tile[(threadIdx.y+(TILE_W - T + 1))*(TILE_W)*D+(threadIdx.x)*D+d]=Dm[ym+xm+d];
			}
			if(((ty+(TILE_W - T + 1))<W&&(threadIdx.y+(TILE_W - T + 1))<(TILE_W))&&((tx+(TILE_W - T + 1))<W&&(threadIdx.x+(TILE_W - T + 1))<(TILE_W)))
			{
				ym=(ty+(TILE_W - T + 1))*W*D;
				xm=(tx+(TILE_W - T + 1))*D;
				tile[(threadIdx.y+(TILE_W - T + 1))*(TILE_W)*D+(threadIdx.x+(TILE_W - T + 1))*D+d]=Dm[ym+xm+d];
			}
		}
	__syncthreads();


	if(ty%STRIDE_LENGTH == 0 && tx%STRIDE_LENGTH == 0)
	{
		float sum=0.0;
		for(int i=0;i<T;i++)
		{
			int yk1=i*T*D;
			int ym1=(threadIdx.y+i)*(TILE_W)*D;
			for(int j=0;j<T;j++)
			{
				int xk1=j*D;
				int xm1=(threadIdx.x+j)*D;
				for(int d=0;d<D;d++){
					sum+=tile[ym1+xm1+d]*ker[yk1+xk1+d];
				}
			}
		}
		if(tx<OWS&&ty<OWS){ 
			Do[bz*OW*OW+(ty/STRIDE_LENGTH)*OW+(tx/STRIDE_LENGTH)]=sum;
		}
	}
}

int main()
{
	//allocating memory for matrix and kernel on the host
	unsigned char *matrix=(unsigned char*)malloc(sizeof(unsigned char)*W*W*D);
	float *kernel=(float*)malloc(sizeof(float)*T*T*D*N);
	float *output=(float *)malloc(sizeof(float)*N*OW*OW);

	//filling matrix and kernel 
	fillMatrix(matrix);
	fillKernel(kernel);

	//allocating memory for the kernel and matrix on the GPU
	unsigned char *Dmatrix;hipMalloc(&Dmatrix,sizeof(unsigned char)*W*W*D);
	float *Dkernel;hipMalloc(&Dkernel,sizeof(float)*N*T*T*D);
	float *Doutput;hipMalloc(&Doutput,sizeof(float)*N*OW*OW);
	int blockdimx=(TILE_W - T + 1);
	int blockdimy=(TILE_W - T + 1);
	int griddimz=N;
	int griddimy=(OWS+blockdimx-1)/blockdimx;
	int griddimx=(OWS+blockdimy-1)/blockdimy;
	dim3 blocks(griddimx, griddimy, griddimz);
	dim3 thrds_per_block(blockdimx, blockdimy);

	//copying matrix and kernel to the GPU
	hipMemcpy(Dmatrix, matrix, sizeof(unsigned char)*W*W*D,hipMemcpyHostToDevice);
	hipMemcpy(Dkernel, kernel, sizeof(float)*T*T*D*N,hipMemcpyHostToDevice);

	//cuda events to time kernel
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;
	hipEventRecord(start,0);

	//cuda kernel call
	conv<<<blocks,thrds_per_block>>>(Dmatrix, Dkernel, Doutput);
	CUDA_CALL(hipGetLastError());
	
	hipDeviceSynchronize();

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f\n",milliseconds);


	hipMemcpy(output, Doutput, sizeof(float)*N*OW*OW,hipMemcpyDeviceToHost);

	//Use print_matrix_to_file function only 
	printtofile(output);

}
